#include "hip/hip_runtime.h"
#include "cudaTools.h"
#include "Indice2D.h"

#include "IndiceTools_GPU.h"
#include "Device.h"
#include "Sphere.h"
#include "RayTracingMath.h"

#include "length.h"

using namespace gpu;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

// Déclaration constante globale
__constant__ Sphere TAB_CM[LENGTH_CM];

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/
/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/
/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/**
 * Call once by the host
 */
__host__ void uploadToCM(Sphere* ptrTabSphere)
    {
    size_t size = LENGTH_CM * sizeof(Sphere);
    int offset = 0;
    HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(TAB_CM), ptrTabSphere, size, offset, hipMemcpyHostToDevice));
    }


__device__ void work(uchar4* ptrDevPixels, Sphere* ptrDevTabSphere, int nbSphere, uint w, uint h, float t, const int TID, const int NB_THREAD)
    {
    RayTracingMath raytracingMath = RayTracingMath(ptrDevTabSphere, nbSphere);

    const int WH = w*h;

    int i;
    int j;

    int s = TID;
    while(s < WH)
	{
	IndiceTools::toIJ(s, w, &i, &j);
	raytracingMath.colorIJ(&ptrDevPixels[s], i, j, t);
	s += NB_THREAD;
	}
    }

__global__ void raytracingGM(uchar4* ptrDevPixels, uint w, uint h, float t, Sphere* ptrDevTabSphere,int tabSphereLength)
    {
    const int TID = Indice2D::tid();
    const int NB_THREAD = Indice2D::nbThread();

    work(ptrDevPixels, ptrDevTabSphere, tabSphereLength, w, h, t, TID, NB_THREAD);
    }

__global__ void raytracingCM(uchar4* ptrDevPixels, uint w, uint h, float t)
    {
    const int TID = Indice2D::tid();
    const int NB_THREAD = Indice2D::nbThread();

    work(ptrDevPixels, TAB_CM, LENGTH_CM, w, h, t, TID, NB_THREAD);
    }

__global__ void raytracingSM(uchar4* ptrDevPixels, uint w, uint h, float t, Sphere* ptrDevTabSphereGM, int tabSphereLength)
    {
    const int TID_LOCAL = Indice2D::tidLocal();
    const int NB_THREAD_LOCAL = Indice2D::nbThreadLocal();

    extern __shared__ Sphere ptrDevTabSphereSM[];

    int s = TID_LOCAL;
    while(s < tabSphereLength)
	{
	ptrDevTabSphereSM[s] = ptrDevTabSphereGM[s];
	s++;
	}

    work(ptrDevPixels, ptrDevTabSphereSM, tabSphereLength, w, h, t, TID_LOCAL, NB_THREAD_LOCAL);
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

